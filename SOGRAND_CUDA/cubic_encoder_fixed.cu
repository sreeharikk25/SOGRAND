#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define THREADS_PER_BLOCK 128
#define CHECK_CUDA(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                hipGetErrorString(error)); \
        exit(1); \
    } \
} while(0)

// Generator matrix in constant memory
__constant__ int d_G[10*15];

// 3D tensor access - both host and device versions
__host__ __device__ inline int tensor_idx(int i, int j, int k, int n) {
    return k * n * n + j * n + i;
}

// Kernel for encoding rows (stage 1)
__global__ void encode_rows_cubic_kernel(int* codeword, int n, int k, int num_blocks) {
    int block_id = blockIdx.z;
    int slice = blockIdx.y;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (block_id >= num_blocks || slice >= k || row >= k) return;
    
    int offset = block_id * n * n * n;
    
    // Extract row data
    int temp_vec[10];  // Max k = 10
    for (int j = 0; j < k; j++) {
        temp_vec[j] = codeword[offset + tensor_idx(row, j, slice, n)];
    }
    
    // Encode parity columns
    for (int col = k; col < n; col++) {
        int sum = 0;
        for (int j = 0; j < k; j++) {
            sum += temp_vec[j] * d_G[j * n + col];
        }
        codeword[offset + tensor_idx(row, col, slice, n)] = sum % 2;
    }
}

// Kernel for encoding columns (stage 2)
__global__ void encode_columns_cubic_kernel(int* codeword, int n, int k, int num_blocks) {
    int block_id = blockIdx.z;
    int slice = blockIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (block_id >= num_blocks || slice >= k || col >= n) return;
    
    int offset = block_id * n * n * n;
    
    // Extract column data
    int temp_vec[10];
    for (int i = 0; i < k; i++) {
        temp_vec[i] = codeword[offset + tensor_idx(i, col, slice, n)];
    }
    
    // Encode parity rows
    for (int row = k; row < n; row++) {
        int sum = 0;
        for (int j = 0; j < k; j++) {
            sum += temp_vec[j] * d_G[j * n + row];
        }
        codeword[offset + tensor_idx(row, col, slice, n)] = sum % 2;
    }
}

// Kernel for encoding slices (stage 3)
__global__ void encode_slices_cubic_kernel(int* codeword, int n, int k, int num_blocks) {
    int block_id = blockIdx.z;
    int row = blockIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (block_id >= num_blocks || row >= n || col >= n) return;
    
    int offset = block_id * n * n * n;
    
    // Extract slice vector
    int temp_vec[10];
    for (int i = 0; i < k; i++) {
        temp_vec[i] = codeword[offset + tensor_idx(row, col, i, n)];
    }
    
    // Encode parity slices
    for (int slice = k; slice < n; slice++) {
        int sum = 0;
        for (int j = 0; j < k; j++) {
            sum += temp_vec[j] * d_G[j * n + slice];
        }
        codeword[offset + tensor_idx(row, col, slice, n)] = sum % 2;
    }
}

// FIXED: Kernel to copy systematic part with correct bit ordering
__global__ void copy_systematic_kernel_fixed(int* input, int* output, int n, int k, int num_blocks) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_elements = num_blocks * k * k * k;
    
    if (idx >= total_elements) return;
    
    int block_id = idx / (k * k * k);
    int local_idx = idx % (k * k * k);
    
    // FIXED: Correct mapping to match C code tensor layout
    // In C: tensor.data[k*dim1*dim2 + j*dim1 + i] for (i,j,k) position
    // So for k×k×k tensor: index = slice*k*k + col*k + row
    int row = local_idx % k;
    int col = (local_idx / k) % k;
    int slice = local_idx / (k * k);
    
    int in_offset = block_id * k * k * k;
    int out_offset = block_id * n * n * n;
    
    output[out_offset + tensor_idx(row, col, slice, n)] = input[in_offset + local_idx];
}

// Host-side CRC matrix generation
int* koopman2matlab(const char* k_poly, int* poly_len) {
    long long dec_val = strtoll(k_poly, NULL, 16);
    int len = (dec_val > 0) ? floor(log2(dec_val)) + 1 : 1;
    *poly_len = len + 1;
    int* poly = (int*)malloc(sizeof(int) * (*poly_len));
    for (int i = 0; i < len; i++) poly[i] = (dec_val >> (len - 1 - i)) & 1;
    poly[len] = 1;
    return poly;
}

void getGH_sys_CRC(int n, int k, int* G_flat) {
    const char* hex_poly = NULL;
    int r = n - k;

    if (r == 3) hex_poly = "0x5";
    else if (r == 4) hex_poly = "0x9";
    else if (r == 5 && k <= 10) hex_poly = "0x15";
    else if (r == 5 && k <= 26) hex_poly = "0x12";
    else if (r == 6 && k <= 25) hex_poly = "0x23";
    else if (r == 6 && k <= 57) hex_poly = "0x33";
    else {
        fprintf(stderr, "Error: (n, k) = (%d, %d) is not supported.\n", n, k);
        exit(1);
    }

    int poly_len;
    int* poly = koopman2matlab(hex_poly, &poly_len);

    // Temporary matrices
    int** P = (int**)malloc(k * sizeof(int*));
    for(int i=0; i<k; ++i) P[i] = (int*)calloc(r, sizeof(int));
    int* msg_poly = (int*)calloc(k + r, sizeof(int));

    // Generate parity matrix P
    for (int i = 0; i < k; i++) {
        memset(msg_poly, 0, (k + r) * sizeof(int));
        msg_poly[i] = 1;

        // Polynomial division
        for (int j = 0; j < k; j++) {
            if (msg_poly[j] == 1) {
                for (int l = 0; l < poly_len; l++) {
                    msg_poly[j + l] ^= poly[l];
                }
            }
        }
        
        // Extract remainder (parity bits)
        for (int j = 0; j < r; j++) P[i][j] = msg_poly[k + j];
    }

    // Build G matrix: G = [I_k | P]
    for (int i = 0; i < k; i++) {
        for (int j = 0; j < k; j++) {
            G_flat[i * n + j] = (i == j) ? 1 : 0;
        }
        for (int j = 0; j < r; j++) {
            G_flat[i * n + (k + j)] = P[i][j];
        }
    }

    // Cleanup
    free(poly);
    free(msg_poly);
    for(int i=0; i<k; ++i) free(P[i]);
    free(P);
}

// Host function to encode multiple blocks
void encode_cubic_cuda_fixed(unsigned char* h_input, unsigned char* h_output, 
                            int num_blocks, int n, int k) {
    int message_block_size = k * k * k;
    int codeword_block_size = n * n * n;
    
    // Convert bytes to bits
    int* h_input_bits = (int*)malloc(num_blocks * message_block_size * sizeof(int));
    int* h_output_bits = (int*)malloc(num_blocks * codeword_block_size * sizeof(int));
    
    // Convert input bytes to bits
    for (int b = 0; b < num_blocks; b++) {
        for (int i = 0; i < message_block_size; i++) {
            int byte_idx = i / 8;
            int bit_idx = 7 - (i % 8);
            h_input_bits[b * message_block_size + i] = 
                (h_input[b * ((message_block_size + 7) / 8) + byte_idx] >> bit_idx) & 1;
        }
    }
    
    // Allocate device memory
    int *d_input, *d_codeword;
    CHECK_CUDA(hipMalloc(&d_input, num_blocks * message_block_size * sizeof(int)));
    CHECK_CUDA(hipMalloc(&d_codeword, num_blocks * codeword_block_size * sizeof(int)));
    
    // Initialize codeword to zero
    CHECK_CUDA(hipMemset(d_codeword, 0, num_blocks * codeword_block_size * sizeof(int)));
    
    // Copy input to device
    CHECK_CUDA(hipMemcpy(d_input, h_input_bits, 
                          num_blocks * message_block_size * sizeof(int),
                          hipMemcpyHostToDevice));
    
    // Copy systematic part with fixed kernel
    int threads = 256;
    int blocks = (num_blocks * message_block_size + threads - 1) / threads;
    copy_systematic_kernel_fixed<<<blocks, threads>>>(d_input, d_codeword, n, k, num_blocks);
    CHECK_CUDA(hipDeviceSynchronize());
    
    // Setup grid dimensions for encoding
    dim3 threadsPerBlock(16);
    dim3 blocksRows((k + threadsPerBlock.x - 1) / threadsPerBlock.x, k, num_blocks);
    dim3 blocksCols((n + threadsPerBlock.x - 1) / threadsPerBlock.x, k, num_blocks);
    dim3 blocksSlices((n + threadsPerBlock.x - 1) / threadsPerBlock.x, n, num_blocks);
    
    // Stage 1: Encode rows
    encode_rows_cubic_kernel<<<blocksRows, threadsPerBlock>>>(d_codeword, n, k, num_blocks);
    CHECK_CUDA(hipDeviceSynchronize());
    
    // Stage 2: Encode columns
    encode_columns_cubic_kernel<<<blocksCols, threadsPerBlock>>>(d_codeword, n, k, num_blocks);
    CHECK_CUDA(hipDeviceSynchronize());
    
    // Stage 3: Encode slices
    encode_slices_cubic_kernel<<<blocksSlices, threadsPerBlock>>>(d_codeword, n, k, num_blocks);
    CHECK_CUDA(hipDeviceSynchronize());
    
    // Copy result back
    CHECK_CUDA(hipMemcpy(h_output_bits, d_codeword,
                          num_blocks * codeword_block_size * sizeof(int),
                          hipMemcpyDeviceToHost));
    
    // Convert bits back to bytes
    for (int b = 0; b < num_blocks; b++) {
        int bit_count = 0;
        unsigned char byte = 0;
        for (int i = 0; i < codeword_block_size; i++) {
            byte = (byte << 1) | h_output_bits[b * codeword_block_size + i];
            bit_count++;
            if (bit_count == 8) {
                h_output[b * ((codeword_block_size + 7) / 8) + (i / 8)] = byte;
                byte = 0;
                bit_count = 0;
            }
        }
        if (bit_count > 0) {
            byte <<= (8 - bit_count);
            h_output[b * ((codeword_block_size + 7) / 8) + (codeword_block_size / 8)] = byte;
        }
    }
    
    // Free memory
    free(h_input_bits);
    free(h_output_bits);
    CHECK_CUDA(hipFree(d_input));
    CHECK_CUDA(hipFree(d_codeword));
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <input_file> <output_file>\n", argv[0]);
        return 1;
    }

    const char* input_filename = argv[1];
    const char* output_filename = argv[2];

    const int n = 15;
    const int k = 10;
    const int message_block_size = k * k * k;
    const int codeword_block_size = n * n * n;

    // Generate CRC matrix
    int h_G[10*15];
    getGH_sys_CRC(n, k, h_G);
    
    // Copy to constant memory
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_G), h_G, sizeof(h_G)));

    FILE* fin = fopen(input_filename, "rb");
    if (!fin) { perror("Error opening input file"); return 1; }
    FILE* fout = fopen(output_filename, "wb");
    if (!fout) { perror("Error opening output file"); fclose(fin); return 1; }

    printf("CUDA Cubic Encoder Fixed (n=%d, k=%d)...\n", n, k);

    // Process in batches
    const int BATCH_SIZE = 64;
    int message_bytes_per_block = (message_block_size + 7) / 8;
    int codeword_bytes_per_block = (codeword_block_size + 7) / 8;
    
    unsigned char* input_buffer = (unsigned char*)malloc(BATCH_SIZE * message_bytes_per_block);
    unsigned char* output_buffer = (unsigned char*)malloc(BATCH_SIZE * codeword_bytes_per_block);
    
    int total_blocks = 0;
    
    while (true) {
        int blocks_read = 0;
        
        // Read blocks
        for (int i = 0; i < BATCH_SIZE; i++) {
            size_t bytes_read = fread(&input_buffer[i * message_bytes_per_block], 
                                     1, message_bytes_per_block, fin);
            if (bytes_read == 0) break;
            
            // Pad if necessary
            if (bytes_read < message_bytes_per_block) {
                memset(&input_buffer[i * message_bytes_per_block + bytes_read], 
                       0, message_bytes_per_block - bytes_read);
            }
            blocks_read++;
        }
        
        if (blocks_read == 0) break;
        
        // Encode batch
        encode_cubic_cuda_fixed(input_buffer, output_buffer, blocks_read, n, k);
        
        // Write output
        fwrite(output_buffer, codeword_bytes_per_block, blocks_read, fout);
        
        total_blocks += blocks_read;
    }

    printf("Encoding complete. %d block(s) encoded.\n", total_blocks);

    free(input_buffer);
    free(output_buffer);
    fclose(fin);
    fclose(fout);

    return 0;
}