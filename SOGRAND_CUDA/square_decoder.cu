#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <thrust/sort.h>
#include <thrust/device_vector.h>

#define THREADS_PER_BLOCK 256
#define MAX_LIST_SIZE 4
#define CHECK_CUDA(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        fprintf(stderr, "CUDA error at %s:%d - %s\n", __FILE__, __LINE__, \
                hipGetErrorString(error)); \
        exit(1); \
    } \
} while(0)

// Constants
__constant__ int d_G[25*31];  // Generator matrix
__constant__ int d_H[6*31];   // Parity check matrix

// Structure for SOGRAND state
struct SOGRANDState {
    double llr[31];
    double absL[31];
    int perm[31];
    uint8_t cHD[31];
    uint8_t c[31];
    uint8_t TEP[31];
    double chat_list[31 * MAX_LIST_SIZE];
    double s_list[4 * MAX_LIST_SIZE];
    double APP_list[MAX_LIST_SIZE];
    int curL;
    double T;
    double pNL;
};

// CUDA kernel for hard decision
__device__ void hard_decision_cuda(double* llr, uint8_t* c, int n) {
    for (int i = 0; i < n; i++) {
        c[i] = (llr[i] > 0.0) ? 0 : 1;
    }
}

// CUDA kernel for parity check - FIXED: matches C implementation
__device__ bool parity_check_cuda(uint8_t* c, int n, int s) {
    for (int j = 0; j < s; j++) {
        uint8_t syndrome = 0;
        for (int i = 0; i < n; i++) {
            syndrome ^= (c[i] * d_H[j*n + i]);
        }
        if (syndrome == 1) return false;  // Fixed: matches C code logic
    }
    return true;
}

// CUDA kernel for parity calculation
__device__ int parity_cuda(uint8_t array[], int n) {
    int sum = 0;
    for (int i = 0; i < n; i++) sum += array[i];
    return sum % 2;
}

// CUDA kernel for JacLog function
__device__ double JacLog_cuda(double x) {
    if (x > 50) return x;
    if (x < -50) return 0.0;
    return log(1.0 + exp(x));
}

// CUDA kernel for probability parity calculation
__device__ double prob_parity_cuda(int parity_cHD, double *absL, int n) {
    double prob_even = 1.0;
    for (int i = 0; i < n; i++) {
        prob_even *= (1.0 - 2.0 * exp(-absL[i]) / (1.0 + exp(-absL[i])));
    }
    prob_even = 0.5 * (1.0 + prob_even);
    return (parity_cHD == 0) ? prob_even : 1.0 - prob_even;
}

// CUDA kernel for AddTEP function
__device__ void AddTEP_cuda(uint8_t *c, uint8_t *cHD, uint8_t *TEP, int *perm, int n) {
    for (int i = 0; i < n; i++) c[perm[i]] = cHD[perm[i]] ^ TEP[i];
}

// CUDA kernel for getPM_HD function
__device__ double getPM_HD_cuda(double *absL, int n) {
    double pm = 0;
    for(int i = 0; i < n; i++) pm += JacLog_cuda(-absL[i]);
    return pm;
}

// CUDA kernel for getPM function
__device__ double getPM_cuda(uint8_t *TEP, double *absL, double PM_HD, int n) {
    double pm = PM_HD;
    for(int i = 0; i < n; i++) {
        if (TEP[i] == 1) pm += (JacLog_cuda(absL[i]) - JacLog_cuda(-absL[i]));
    }
    return pm;
}

// CUDA kernel for getLConf function
__device__ double getLConf_cuda(double *pNL, double P_notGuess, int cur_L, double *score, int s, int even) {
    double P_positive = 0.0;
    for(int i = 0; i < cur_L; i++) P_positive += exp(-score[4*i+1]);

    if(even == 1) s--;

    double P_negative = pow(2.0, -(double)s) * P_notGuess;
    pNL[0] = P_negative;
    return (P_positive + P_negative > 1e-9) ? (P_positive / (P_positive + P_negative)) : 1.0;
}

// CUDA kernel for mountain_build function
__device__ void mountain_build_cuda(int *u, int k, int w, int W1, int n1) {
    for(int i = k + 1; i < w; i++) u[i] = u[k];
    long long W2 = W1;
    for(int i = 0; i < w; i++) W2 -= u[i];

    if (n1 - u[k] == 0) return;

    unsigned long long q = (W2 >= 0) ? (unsigned long long)floor((double)W2 / (n1 - u[k])) : 0;
    unsigned long long r = W2 - q * (n1 - u[k]);
    if (q > 0 && w > q) {
        for(int i = w - q; i < w; i++) u[i] = n1;
    }
    if (w > q) u[w - q - 1] += r;
}

// CUDA kernel for getAPP function
__device__ void getAPP_cuda(int cur_L, double *score, double *APP) {
    if (cur_L == 0) return;
    double P_positive = 0.0;
    for(int i = 0; i < cur_L; i++) P_positive += exp(-score[4*i+1]);
    if (P_positive < 1e-30) return;
    double denominator = score[4*(cur_L-1)+3] / P_positive;
    for(int i = 0; i < cur_L; i++) APP[i] = exp(-score[4*i+1]) * denominator;
}

// CUDA kernel for findMax function
__device__ int findMax_cuda(int a, int b) { 
    return !(b > a) ? a : b; 
}

// Improved SOGRAND kernel with proper algorithm implementation
__device__ void sogrand_siso_cuda(double* L_APP, double* L_E, double* llr,
                                  int n, int k, SOGRANDState* state, int even, double thres) {
    // Initialize
    hard_decision_cuda(llr, state->cHD, n);
    int parity_cHD = parity_cuda(state->cHD, n);
    state->pNL = 0.0;

    // Calculate absolute LLRs and initialize permutation
    for (int i = 0; i < n; i++) {
        state->absL[i] = fabs(llr[i]);
        state->perm[i] = i;
        state->TEP[i] = 0;
    }

    // Simple bubble sort for reliability ordering (can be optimized)
    for (int i = 0; i < n-1; i++) {
        for (int j = 0; j < n-i-1; j++) {
            if (state->absL[j] > state->absL[j+1]) {
                double temp = state->absL[j];
                state->absL[j] = state->absL[j+1];
                state->absL[j+1] = temp;
                int temp_perm = state->perm[j];
                state->perm[j] = state->perm[j+1];
                state->perm[j+1] = temp_perm;
            }
        }
    }

    // Initialize state
    for (int i = 0; i < n; i++) {
        state->c[i] = state->cHD[i];
    }
    state->curL = 0;
    state->T = 1;

    // Initialize score array
    for (int i = 0; i < 4 * MAX_LIST_SIZE; i++) {
        state->s_list[i] = 0;
    }
    for (int i = 0; i < MAX_LIST_SIZE; i++) {
        state->APP_list[i] = 0;
    }

    double P_notGuess = 1.0;
    if (even == 1) {
        P_notGuess = prob_parity_cuda(parity_cHD, state->absL, n);
    }

    double PM_HD = getPM_HD_cuda(state->absL, n);

    // Check hard decision first
    AddTEP_cuda(state->c, state->cHD, state->TEP, state->perm, n);
    if (parity_cHD == 0 || even == 0) {
        P_notGuess -= exp(-getPM_cuda(state->TEP, state->absL, PM_HD, n));
    }

    if (parity_check_cuda(state->c, n, n-k)) {
        state->s_list[1] = getPM_cuda(state->TEP, state->absL, PM_HD, n);
        for (int i = 0; i < n; i++) {
            state->chat_list[i] = state->c[i];
        }
        state->s_list[2] = 1;
        state->s_list[3] = getLConf_cuda(&state->pNL, P_notGuess, state->curL, state->s_list, n-k, even);
        state->curL++;
        
        if ((state->s_list[3] > thres) || (state->curL == MAX_LIST_SIZE)) {
            getAPP_cuda(state->curL, state->s_list, state->APP_list);
            // Compute final L_APP and L_E
            for (int i = 0; i < n; i++) {
                double p0 = 0, p1 = 0;
                for (int l = 0; l < state->curL; l++) {
                    if (state->chat_list[l * n + i] == 1) {
                        p1 += state->APP_list[l];
                    } else {
                        p0 += state->APP_list[l];
                    }
                }
                L_APP[i] = log(fmax(p0, 1e-30)) - log(fmax(p1, 1e-30));
                L_E[i] = L_APP[i] - llr[i];
            }
            return;
        }
    }

    // Simplified TEP generation (limited due to GPU constraints)
    // In a full implementation, this would use the mountain climbing algorithm
    int max_flips = min(4, n);
    for (int w = 1; w <= max_flips && state->curL < MAX_LIST_SIZE; w++) {
        if (even == 1 && (w % 2 != parity_cHD)) continue;
        
        // Generate simple TEP patterns
        for (int i = 0; i < n; i++) state->TEP[i] = 0;
        for (int i = 0; i < w; i++) state->TEP[i] = 1;

        AddTEP_cuda(state->c, state->cHD, state->TEP, state->perm, n);
        state->T++;
        P_notGuess -= exp(-getPM_cuda(state->TEP, state->absL, PM_HD, n));

        if (parity_check_cuda(state->c, n, n-k)) {
            state->s_list[4*state->curL] = w;
            state->s_list[4*state->curL+1] = getPM_cuda(state->TEP, state->absL, PM_HD, n);
            for (int i = 0; i < n; i++) {
                state->chat_list[state->curL * n + i] = state->c[i];
            }
            state->s_list[4*state->curL+2] = state->T;
            state->s_list[4*state->curL+3] = getLConf_cuda(&state->pNL, P_notGuess, state->curL, state->s_list, n-k, even);
            state->curL++;
            
            if ((state->s_list[4*(state->curL-1)+3] > thres) || (state->curL == MAX_LIST_SIZE)) {
                getAPP_cuda(state->curL, state->s_list, state->APP_list);
                // Compute final L_APP and L_E
                for (int i = 0; i < n; i++) {
                    double p0 = 0, p1 = 0;
                    for (int l = 0; l < state->curL; l++) {
                        if (state->chat_list[l * n + i] == 1) {
                            p1 += state->APP_list[l];
                        } else {
                            p0 += state->APP_list[l];
                        }
                    }
                    L_APP[i] = log(fmax(p0, 1e-30)) - log(fmax(p1, 1e-30));
                    L_E[i] = L_APP[i] - llr[i];
                }
                return;
            }
        }
    }

    // If no valid codewords found, use channel LLRs
    if (state->curL == 0) {
        for (int i = 0; i < n; i++) {
            L_APP[i] = llr[i];
            L_E[i] = 0;
        }
    } else {
        getAPP_cuda(state->curL, state->s_list, state->APP_list);
        // Compute final L_APP and L_E
        for (int i = 0; i < n; i++) {
            double p0 = 0, p1 = 0;
            for (int l = 0; l < state->curL; l++) {
                if (state->chat_list[l * n + i] == 1) {
                    p1 += state->APP_list[l];
                } else {
                    p0 += state->APP_list[l];
                }
            }
            L_APP[i] = log(fmax(p0, 1e-30)) - log(fmax(p1, 1e-30));
            L_E[i] = L_APP[i] - llr[i];
        }
    }
}

// Kernel for row decoding with reduced shared memory usage
__global__ void decode_rows_kernel(double* L_channel, double* L_APP, double* L_E,
                                   double alpha, int n, int k, int num_blocks) {
    int block_id = blockIdx.y;
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (block_id >= num_blocks || row >= n) return;

    __shared__ SOGRANDState states[8];

    int offset = block_id * n * n;
    SOGRANDState* state = &states[threadIdx.x % 8];

    // Prepare input
    double input[31];
    for (int col = 0; col < n; col++) {
        int idx = offset + row * n + col;
        input[col] = L_channel[idx] + alpha * L_E[idx];
    }

    // Run SOGRAND
    double L_APP_row[31], L_E_row[31];
    sogrand_siso_cuda(L_APP_row, L_E_row, input, n, k, state, 1, 0.999); // even=1, thres=0.999

    // Write results
    for (int col = 0; col < n; col++) {
        int idx = offset + row * n + col;
        L_APP[idx] = L_APP_row[col];
        L_E[idx] = L_E_row[col];
    }
}

// Kernel for column decoding with reduced shared memory usage
__global__ void decode_columns_kernel(double* L_channel, double* L_APP, double* L_E,
                                     double alpha, int n, int k, int num_blocks) {
    int block_id = blockIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (block_id >= num_blocks || col >= n) return;

    __shared__ SOGRANDState states[8];

    int offset = block_id * n * n;
    SOGRANDState* state = &states[threadIdx.x % 8];

    // Prepare input
    double input[31];
    for (int row = 0; row < n; row++) {
        int idx = offset + row * n + col;
        input[row] = L_channel[idx] + alpha * L_E[idx];
    }

    // Run SOGRAND
    double L_APP_col[31], L_E_col[31];
    sogrand_siso_cuda(L_APP_col, L_E_col, input, n, k, state, 1, 0.999); // even=1, thres=0.999

    // Write results
    for (int row = 0; row < n; row++) {
        int idx = offset + row * n + col;
        L_APP[idx] = L_APP_col[row];
        L_E[idx] = L_E_col[row];
    }
}

// Host function for square decoding
void decode_square_cuda(double* h_llr_buffer, int* h_bit_buffer, int num_blocks,
                       int n, int k, int Imax, double* alpha) {
    size_t matrix_size = n * n * sizeof(double);

    // Allocate device memory
    double *d_L_channel, *d_L_APP, *d_L_E;

    CHECK_CUDA(hipMalloc(&d_L_channel, num_blocks * matrix_size));
    CHECK_CUDA(hipMalloc(&d_L_APP, num_blocks * matrix_size));
    CHECK_CUDA(hipMalloc(&d_L_E, num_blocks * matrix_size));

    // Copy input to device
    CHECK_CUDA(hipMemcpy(d_L_channel, h_llr_buffer, num_blocks * matrix_size,
                          hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemset(d_L_E, 0, num_blocks * matrix_size));

    // Setup grid dimensions
    dim3 threadsPerBlock(32);
    dim3 blocksPerGrid((n + threadsPerBlock.x - 1) / threadsPerBlock.x, num_blocks);

    // Iterative decoding
    for (int iter = 0; iter < Imax; iter++) {
        // Decode rows
        decode_rows_kernel<<<blocksPerGrid, threadsPerBlock>>>(
            d_L_channel, d_L_APP, d_L_E, alpha[2*iter], n, k, num_blocks);
        CHECK_CUDA(hipDeviceSynchronize());

        // Decode columns
        decode_columns_kernel<<<blocksPerGrid, threadsPerBlock>>>(
            d_L_channel, d_L_APP, d_L_E, alpha[2*iter+1], n, k, num_blocks);
        CHECK_CUDA(hipDeviceSynchronize());
    }

    // Extract message bits
    double* h_L_APP = (double*)malloc(num_blocks * matrix_size);
    CHECK_CUDA(hipMemcpy(h_L_APP, d_L_APP, num_blocks * matrix_size,
                          hipMemcpyDeviceToHost));

    // Hard decision to get bits
    for (int b = 0; b < num_blocks; b++) {
        for (int i = 0; i < k; i++) {
            for (int j = 0; j < k; j++) {
                int llr_idx = b * n * n + i * n + j;
                int bit_idx = b * k * k + i * k + j;
                h_bit_buffer[bit_idx] = (h_L_APP[llr_idx] > 0) ? 0 : 1;
            }
        }
    }

    free(h_L_APP);
    CHECK_CUDA(hipFree(d_L_channel));
    CHECK_CUDA(hipFree(d_L_APP));
    CHECK_CUDA(hipFree(d_L_E));
}

// CRC polynomial conversion
int* koopman2matlab(const char* k_poly, int* poly_len) {
    long long dec_val = strtoll(k_poly, NULL, 16);
    int len = (dec_val > 0) ? floor(log2(dec_val)) + 1 : 1;
    *poly_len = len + 1;
    int* poly = (int*)malloc(sizeof(int) * (*poly_len));
    for (int i = 0; i < len; i++) {
        poly[i] = (dec_val >> (len - 1 - i)) & 1;
    }
    poly[len] = 1;
    return poly;
}

// Proper matrix initialization for square code (copied from encoder)
void init_matrices_square(int* G_flat, int* H_flat, int n, int k) {
    const char* hex_poly = NULL;
    int r = n - k;

    if (r == 3) hex_poly = "0x5";
    else if (r == 4) hex_poly = "0x9";
    else if (r == 5 && k <= 10) hex_poly = "0x15";
    else if (r == 5 && k <= 26) hex_poly = "0x12";
    else if (r == 6 && k <= 25) hex_poly = "0x23";
    else if (r == 6 && k <= 57) hex_poly = "0x33";
    else {
        fprintf(stderr, "Error: (n, k) = (%d, %d) is not supported.\n", n, k);
        exit(1);
    }

    int poly_len;
    int* poly = koopman2matlab(hex_poly, &poly_len);

    // Generate parity matrix P
    int** P = (int**)malloc(k * sizeof(int*));
    for(int i = 0; i < k; i++) P[i] = (int*)malloc(r * sizeof(int));
    int* msg_poly = (int*)calloc(k + r, sizeof(int));

    for (int i = 0; i < k; i++) {
        memset(msg_poly, 0, (k + r) * sizeof(int));
        msg_poly[i] = 1;

        for (int j = 0; j < k; j++) {
            if (msg_poly[j] == 1) {
                for (int l = 0; l < poly_len; l++) {
                    msg_poly[j + l] ^= poly[l];
                }
            }
        }
        for (int j = 0; j < r; j++) P[i][j] = msg_poly[k + j];
    }

    // Build generator matrix G = [I_k | P]
    for (int i = 0; i < k; i++) {
        for (int j = 0; j < k; j++) {
            G_flat[i * n + j] = (i == j) ? 1 : 0;
        }
        for (int j = 0; j < r; j++) {
            G_flat[i * n + k + j] = P[i][j];
        }
    }

    // Build parity check matrix H = [P^T | I_r]
    for (int i = 0; i < r; i++) {
        for (int j = 0; j < k; j++) {
            H_flat[i * n + j] = P[j][i];
        }
        for (int j = 0; j < r; j++) {
            H_flat[i * n + k + j] = (i == j) ? 1 : 0;
        }
    }

    // Cleanup
    free(poly);
    free(msg_poly);
    for(int i = 0; i < k; i++) free(P[i]);
    free(P);
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        fprintf(stderr, "Usage: %s <input_llr_file> <output_file>\n", argv[0]);
        return 1;
    }

    const char* input_filename = argv[1];
    const char* output_filename = argv[2];

    const int n = 31;
    const int k = 25;
    const int codeword_block_size = n * n;
    const int message_block_size = k * k;
    const int Imax = 20;

    // Initialize alpha array
    double alpha[50];
    for(int i = 0; i < 50; i++) alpha[i] = 0.5;

    // Setup generator and parity check matrices
    int h_G[25*31], h_H[6*31];
    init_matrices_square(h_G, h_H, n, k);

    // Copy matrices to constant memory
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_G), h_G, sizeof(h_G)));
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_H), h_H, sizeof(h_H)));

    FILE* fin = fopen(input_filename, "rb");
    if (!fin) { perror("Error opening input file"); return 1; }
    FILE* fout = fopen(output_filename, "wb");
    if (!fout) { perror("Error opening output file"); fclose(fin); return 1; }

    printf("CUDA Square Decoder (n=%d, k=%d)...\n", n, k);

    // Process multiple blocks at once for better GPU utilization
    const int BATCH_SIZE = 32;
    double* llr_batch = (double*)malloc(BATCH_SIZE * codeword_block_size * sizeof(double));
    int* bit_batch = (int*)malloc(BATCH_SIZE * message_block_size * sizeof(int));

    int total_blocks = 0;

    while (true) {
        // Read batch of blocks
        int blocks_read = 0;
        for (int i = 0; i < BATCH_SIZE; i++) {
            size_t read = fread(&llr_batch[i * codeword_block_size],
                               sizeof(double), codeword_block_size, fin);
            if (read != codeword_block_size) break;
            blocks_read++;
        }

        if (blocks_read == 0) break;

        // Decode batch on GPU
        decode_square_cuda(llr_batch, bit_batch, blocks_read, n, k, Imax, alpha);

        // Convert bits to bytes and write
        for (int b = 0; b < blocks_read; b++) {
            unsigned char byte_out = 0;
            int bit_count_out = 0;

            for (int i = 0; i < message_block_size; i++) {
                byte_out = (byte_out << 1) | bit_batch[b * message_block_size + i];
                bit_count_out++;
                if (bit_count_out == 8) {
                    fwrite(&byte_out, 1, 1, fout);
                    byte_out = 0;
                    bit_count_out = 0;
                }
            }

            if (bit_count_out > 0) {
                byte_out <<= (8 - bit_count_out);
                fwrite(&byte_out, 1, 1, fout);
            }
        }

        total_blocks += blocks_read;
    }

    printf("Decoding complete. %d block(s) decoded.\n", total_blocks);

    free(llr_batch);
    free(bit_batch);
    fclose(fin);
    fclose(fout);

    return 0;
}
